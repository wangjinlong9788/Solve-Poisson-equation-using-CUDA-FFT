#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <fstream>
#include <math.h>
#include <complex>
#include <hipfft/hipfft.h>
__global__ void solve_poisson(hipfftComplex *ft, hipfftComplex *ft_k, float *k, int N) 
 {  int i = threadIdx.x + blockIdx.x*BSZ;
    int j = threadIdx.y + blockIdx.y*BSZ; 
	int index = j*N+i; 
	if (i<N && j<N)
	{               
	  float k2 = k[i]*k[i]+k[j]*k[j];
	  if (i==0 && j==0) {k2 = 1.0f;} 
	  ft_k[index].x = -ft[index].x/k2;
	  ft_k[index].y = -ft[index].y/k2;
	}
  }

__global__ void real2complex(float *f, hipfftComplex *fc, int N) 
  {          int i = threadIdx.x + blockIdx.x*blockDim.x; 
  int j = threadIdx.y + blockIdx.y*blockDim.y;  
  int index = j*N+i;  
  if (i<N && j<N)   
  {       fc[index].x = f[index];     
         fc[index].y = 0.0f; 
   }  
  }  

__global__ void complex2real(hipfftComplex *fc, float *f, int N)
  {          int i = threadIdx.x + blockIdx.x*BSZ;
  int j = threadIdx.y + blockIdx.y*BSZ; 
  int index = j*N+i;         
  if (i<N && j<N)  
  {   f[index] = fc[index].x/((float)N*(float)N); 
      //divide by number of elements to recover value    
  }
  }

int main()  
{
   int N = 64;  //block number 
   float xmax=1.0f, xmin=0.0f,ymin=0.0f,h=(xmax‐xmin)/((float)N),s=0.1,s2=s*s;   //define interval, sigma
   float *x=new float[N*N],*y=new float[N*N],*u=new float[N*N],*f = new float[N*N],*u_a=new float[N*N],*err =new float[N*N]; //define x,y,u,ua   
   float r2;          
   for (int j=0; j<N; j++)                 
      for (int i=0; i<N; i++)                 
	 { x[N*j+i] = xmin + i*h;  
	   y[N*j+i] = ymin + j*h;                        
           r2 = (x[N*j+i]-0.5)*(x[N*j+i]-0.5) + (y[N*j+i]-0.5)*(y[N*j+i]-0.5);  //define r^2
	   f[N*j+i] = (r2-2*s2)/(s2*s2)*exp(-r2/(2*s2)); //define f at right hand side
	   u_a[N*j+i] = exp(-r2/(2*s2)); // analytical solution     
	 }          
   float   *k = new float[N],M_PI=3.14159;       
   for (int i=0; i<=N/2; i++)          
	{
	   k[i] = i*2*M_PI;
        } 
   for (int i=N/2+1; i<N; i++)          
	 {
           k[i] = (i ‐ N) * 2*M_PI;
	 }
		 
    // Allocate arrays on the device   
	
        float *k_d, *f_d, *u_d; 
        hipMalloc ((void**)&k_d, sizeof(float)*N); 
	hipMalloc ((void**)&f_d, sizeof(float)*N*N); 
	hipMalloc ((void**)&u_d, sizeof(float)*N*N);
	hipMemcpy(k_d, k, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(f_d, f, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipfftComplex *ft_d, *f_dc, *ft_d_k, *u_dc;
	hipMalloc ((void**)&ft_d, sizeof(hipfftComplex)*N*N);
	hipMalloc ((void**)&ft_d_k, sizeof(hipfftComplex)*N*N);
	hipMalloc ((void**)&f_dc, sizeof(hipfftComplex)*N*N); 
	hipMalloc ((void**)&u_dc, sizeof(hipfftComplex)*N*N); 
	dim3 dimGrid  (int((N-0.5)/BSZ) + 1, int((N-0.5)/BSZ) + 1); 
	dim3 dimBlock (BSZ, BSZ); 
	real2complex<<<dimGrid, dimBlock>>>(f_d, f_dc, N);
	hipfftHandle plan;
	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);
	
	hipfftExecC2C(plan, f_dc, ft_d, HIPFFT_FORWARD);
	solve_poisson<<<dimGrid, dimBlock>>>(ft_d, ft_d_k, k_d, N);
	hipfftExecC2C(plan, ft_d_k, u_dc, HIPFFT_BACKWARD);
	complex2real<<<dimGrid, dimBlock>>>(u_dc, u_d, N);
	hipMemcpy(u, u_d, sizeof(float)*N*N, hipMemcpyDeviceToHost); 
	float constant = u[0]; 
	for (int i=0; i<N*N; i++)
	{       
	   u[i] -= constant; //substract u[0] to force the arbitrary constant to be 0
	}
	
	hipFree(k_d);
	hipFree(f_d);
	hipFree(u_d);
	hipFree(ft_d);
	hipFree(f_dc);
	hipFree(ft_d_k);
	hipFree(u_dc);

}
